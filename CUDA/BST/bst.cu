#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "lines.cu"
#include <string.h>
#include <stdio.h>

/*
 * C / CUDA Implementation BST.
 * Modified to discard any duplicate value.
*/


struct pair{
    int a;
    int b;

    __host__ __device__ 
    friend bool operator<= (const pair& l,const pair& r)
    {return (l.a + l.b) <= (r.a + r.b);}

    __host__ __device__ 
    friend bool operator>= (const pair& l,const pair& r)
    {return (l.a + l.b) >= (r.a + r.b);}

    __host__ __device__ 
    friend bool operator !=(const pair& l,const pair& r)
    {return !((l.a == r.a) && (l.b == r.b));}

};


struct node 
{ 
    pair key; 
    struct node *left, *right; 
}; 

__host__ __device__
pair makePair(int l1,int l2)
{
  return {l1,l2};
};

__host__ __device__
struct node *newNode(pair pass) 
{ 
    struct node *temp =  (struct node *)malloc(sizeof(struct node)); 
    temp->key = pass;
    temp->left = temp->right = NULL; 
    return temp; 
} 


   
__host__ __device__ 
void inorder(struct node *root) 
{ 
    if (root != NULL) 
    { 
        inorder(root->left); 
        printf("%i %i\n", root->key.a,root->key.b); 
        inorder(root->right); 
    } 
} 
   
__host__ __device__ 
struct node* insert(struct node* node, pair key) 
{ 
    if (node == NULL) return newNode(key); 
    
    if(key <= node->key && node->key != key)
        node->left  = insert(node->left, key); 
    else if(key >= node->key && node->key != key)
        node->right = insert(node->right, key);    
        
    return node; 
} 

// Driver code 
__global__
void mainDevice() 
{ 
	struct node *root = NULL; 
    root = insert(root, makePair(1,1)); 
    insert(root, makePair(1,2)); 
    insert(root, makePair(1,2)); 
    insert(root, makePair(1,3)); 
    insert(root, makePair(2,1)); 
    insert(root, makePair(2,2)); 
    insert(root, makePair(2,2)); 
    insert(root, makePair(2,3)); 
    insert(root, makePair(3,1)); 
    insert(root, makePair(3,2)); 
   
    // print inoder traversal of the BST 
    inorder(root); 
} 

__host__
int main()
{
     mainDevice<<<1,1>>>();

     hipDeviceSynchronize();
    return 0;
}



// This code is contributed by pkthapa 

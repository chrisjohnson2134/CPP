#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

// #include <stdio.h>

//set datafile separator ","
//plot 'test/testSmall.csv' using 1:2:($3-$1):($4-$2) with vectors nohead, "test/points.csv" pt 7 ps 2
//plot 'test/triangles.csv' using 1:2:($3-$1):($4-$2) with vectors nohead, "test/points.csv" pt 7 ps 2

// struct arg_struct
// {
//   std::string outputPath;
// };



struct Point
{
  double x;
  double y;
};

struct Line
{
  int id;
  Point left_point;
  Point right_point;
};

struct linePair
{
  Line l1;
  Line l2;
};


__device__
Line makeLine(int id, int x1, int y1, int x2, int y2);
// void printLine(Line passLine);
// // bool doIntersect(Point p1, Point q1, Point p2, Point q2);
__device__
bool get_line_intersection(Point p0, Point p1,
                           Point p2, Point p3, double *i_x, double *i_y);
// std::vector<std::string> split(const std::string &str, const std::string &delim);
// void *multiply(void *passed);
Line makeLineHost(int id, int x1, int y1, int x2, int y2);
__device__
linePair makeLinePair(Line l1,Line l2);
// //Share Fields
std::map<std::string, int> mymap;
// int sharedI = 0;

// __device__ int* mutex = 0;

__device__ void lock(int* mutex) {
    /* compare mutex to 0.
       when it equals 0, set it to 1
       we will break out of the loop after mutex gets set to 1 */
    while (atomicCAS(mutex, 0, 1) != 0) {
        /* do nothing */
    }
}

__device__ void unlock(int* mutex) {
    atomicExch(mutex, 0);
}






/////////////////!!!!!!!!!!!!!!!!!!!!!!!!!///////////////////////////////
//USE THE BLOCK AND THREAD to be your for loop INDEXER////////////////////
//////////////////////////////////////////////////////////////////////////
__global__
void runner(Line *myvector,int size,linePair* pairs,int* counter,int* mutex)//,int* points)//, std::string outputPath)
{
  if(blockIdx.x == 0 && threadIdx.x == 0)
  {
    printf("true");
    mutex = 0;
    counter = 0;
  }
  // printf("size : %i \n",size);
//   std::string temp;
  double x, y;
//   std::ofstream omyfile;
  int i = 0;
//   omyfile.open("test/points.csv", std::ios_base::app);

    i = (blockIdx.x *1024) + threadIdx.x;
    //atomicAdd(&counter, 1);
    if(i == 15000)

      printf("thread : %i\n",i );
    for (int j = 0; j < size; j++)
    {
      // if(j == 10)
      //   printf("j : %i\n",j);
      if (i != j)
      {
        if (get_line_intersection(myvector[i].left_point, myvector[i].right_point,
                                  myvector[j].left_point, myvector[j].right_point, &x, &y))
        {
            // (*points)++;
            
          // printf("counter : %i\n",counter);
          //cout << "intersection found." << endl;
          lock(mutex);
          // if (myvector[i].id < myvector[j].id)
          //   pairs[counter] = makeLinePair(myvector[i],myvector[j]);
          // else
          //   pairs[counter] = makeLinePair(myvector[j],myvector[i]);

            // atomicAdd(&counter, 1);
            (*counter)++;
            unlock(mutex);
        //   mtxMap.lock();
        //   if (mymap.find(temp) == mymap.end()) //O(log(n)) disable if you want more speed up I guess
        //   {
            // omyfile << std::to_string(x) << "," << std::to_string(y) << "\n";
            // mymap.insert(std::pair<std::string, int>(temp, 0));
        //   }
        //   mtxMap.unlock();
        }
      }
    }

//   omyfile.close();
}


// void *multiply(void *passed)
// {

//   return NULL;
// }

// Point makePoint(double x, double y)
// {
//   return {x, y};
// }

__device__
Line makeLine(int id, int x1, int y1, int x2, int y2)
{
  Point left_point;
  Point right_point;
  if (x1 < x2)
  {
    left_point.x = x1;
    left_point.y = y1;
    right_point.x = x2;
    right_point.y = y2;
  }
  else
  {
    left_point.x = x2;
    left_point.y = y2;
    right_point.x = x1;
    right_point.y = y1;
  }
  return {id, left_point, right_point};
}

__device__
linePair makeLinePair(Line l1,Line l2)
{
  return {l1,l2};
}

// void printPoint(Point passPoint)
// {
//   printf("x: %f y: %f \n",
//          passPoint.x, passPoint.y);
// }

// void printLine(Line passLine)
// {
//   printf("x1: %f y1: %f :: x2: %f y2: %f",
//          passLine.left_point.x, passLine.left_point.y,
//          passLine.right_point.x, passLine.right_point.y);
// }

// Returns 1 if the lines intersect, otherwise 0. In addition, if the lines
// intersect the intersection point may be stored in the doubles i_x and i_y.
__device__
bool get_line_intersection(Point p0, Point p1,
                           Point p2, Point p3, double *i_x, double *i_y)
{
  // cout << "intersect function" << endl;
  Point s1, s2;
  s1.x = p1.x - p0.x;
  s1.y = p1.y - p0.y;
  s2.x = p3.x - p2.x;
  s2.y = p3.y - p2.y;

  double s, t;
  s = (-s1.y * (p0.x - p2.x) + s1.x * (p0.y - p2.y)) / (-s2.x * s1.y + s1.x * s2.y);
  t = (s2.x * (p0.y - p2.y) - s2.y * (p0.x - p2.x)) / (-s2.x * s1.y + s1.x * s2.y);

  if (s >= 0 && s <= 1 && t >= 0 && t <= 1)
  {
    // Collision detected
    if (i_x != NULL)
      *i_x = p0.x + (t * s1.x);
    if (i_y != NULL)
      *i_y = p0.y + (t * s1.y);
    return true;
  }

  return false; // No collision
}


Line makeLineHost(int id, int x1, int y1, int x2, int y2)
{
  Point left_point;
  Point right_point;
  if (x1 < x2)
  {
    left_point.x = x1;
    left_point.y = y1;
    right_point.x = x2;
    right_point.y = y2;
  }
  else
  {
    left_point.x = x2;
    left_point.y = y2;
    right_point.x = x1;
    right_point.y = y1;
  }
  return {id, left_point, right_point};
}
#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <map>
#include <pthread.h>
#include <fstream>
#include "Intersection.cu"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

using namespace std;

vector<string> split(const string &str, const string &delim);



int main(){

  //struct arg_struct pass;
  int lineNumber = 0;
  vector<Line> localVector;

  string line;
  //string path = "test/testSmall.csv";
  string path = "test/triangles.csv";
  ifstream myfile(path);
  cout << "Lines File Path: " << path << "\n";
  ofstream omyfile;
  omyfile.open("test/points.csv", ofstream::out | ofstream::trunc);
  omyfile.close();
  vector<string> tempSplit;
  if (myfile.is_open())
  {
    while (getline(myfile, line))
    {
      tempSplit = split(line, ",");
      localVector.push_back(makeLineHost(lineNumber,
                                  stod(tempSplit.at(0)), stod(tempSplit.at(1)),
                                  stod(tempSplit.at(2)), stod(tempSplit.at(3))));
      lineNumber++;
    }
    myfile.close();
  }
  string temp;

  // cout << "myvector size : " << localVector.capacity() << "\n";
  localVector.shrink_to_fit();
  cout << "myvector size shrunk: " << localVector.capacity() << "\n";

  //int i = 0;

  const int count = localVector.size();
  const int countLinePair = 40000;
  const int size = count * sizeof(Line);
  const int sizeLinePair = countLinePair * sizeof(linePair);
  printf("pairs : %i\n",sizeLinePair);
  printf("lines : %i\n",size);
  Line *dLines;
  linePair* dpairs;
  hipMalloc(&dpairs,sizeLinePair);
  hipMalloc(&dLines,size);

  Line* hLines = &localVector[0];
  
  hipMemcpy(dLines,hLines,size,hipMemcpyHostToDevice);
//   cout<<hLines[0].left_point.x<<"\n";
//   cout<<hLines2[0].left_point.x<<"\n";
  // cout<<"Blocks : "<<(count/1024)+1;
  int* counter = 0;
  int* mutex = 0;
  runner<<<(count/1024)+1,1024>>>(dLines,count,dpairs,counter,mutex);//,"test/points.csv");

  hipDeviceSynchronize();

  printf("host quite!!");
  hipFree(dLines);
  hipFree(dpairs);
  free(hLines);
  localVector = vector<Line>();
  //i = runner(localVector,"test/points.csv",numCores);
//   int* c = 0;
//   hipMemcpy(counter,c,sizeof(counter),hipMemcpyDeviceToHost);

//   cout << "Number of Intersections Found : " << c << "\n";
}


vector<string> split(const string &str, const string &delim)
{
  std::vector<std::string> tokens;
  size_t prev = 0, pos = 0;
  do
  {
    pos = str.find(delim, prev);
    if (pos == std::string::npos)
      pos = str.length();
    std::string token = str.substr(prev, pos - prev);
    if (!token.empty())
      tokens.push_back(token);
    prev = pos + delim.length();
  } while (pos < str.length() && prev < str.length());
  return tokens;
}

